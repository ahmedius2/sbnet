#include "hip/hip_runtime.h"
/*

   Sparse Blocks Network
   Copyright (c) 2017, Uber Technologies, Inc.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

/*
From GitHub page:

Current code is not tuned for performance with non-square block sizes and has specialized implementations for a specific list of block sizes. This includes square blocks of sizes 1 to 34 and a few others. To achieve maximum performance for these sizes you would need to add your custom template instantiations by modifying SIZE_TEMPLATES macro in sparse_gather.cu
*/

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include "sparse_blocks.cu.h"
#include "cuda_helpers.h"
#include "op_utils.h"

#define COMPUTE_R1(RR) ((RR) < 7 ? ((RR) == 1 ? 1 : 2) : 4)

namespace {
template<typename scalar_t>
struct LaunchParams {
    dim3 block, grid;
    int shmemSize;
    int bSzH1;
    int fittingC1;
    enum { MAX_SHMEM = 24*1024 };
    LaunchParams(int C, int bSzH, int bSzW, int numActive)
    {
        fittingC1 = std::min(32, C);
        bSzH1 = COMPUTE_R1(bSzH);
        while ((shmemSize = (fittingC1+1)*bSzH1*bSzW*sizeof(scalar_t)) > MAX_SHMEM)
            fittingC1--;
        assert(fittingC1 >= 1);
        assert(bSzH1*bSzW*(fittingC1+1)*sizeof(scalar_t) <= MAX_SHMEM);
        block = dim3(512, 1, 1);
        grid = dim3(numActive, DIVUP(C, fittingC1), DIVUP(bSzH, bSzH1));
    }
};
}

// Define the GPU implementation that launches the CUDA kernel.
torch::Tensor LaunchSparseGatherGPU(
        torch::Tensor x, int N, int H, int W, int C, torch::Tensor y,
        int bOffsH0, int bOffsW0, int bSzH, int bSzW, int bStrH, int bStrW,
        int numActive, torch::Tensor activeBlockIndices, bool transpose)
{
	bool hasInst = false;
	hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
#if 1
	#define CALL(RR, CC1, trans) \
		if (bSzH == RR && bSzW == RR && lp.fittingC1 == CC1) { \
			hasInst = true; \
			blockGatherTiled0<scalar_t, 512, RR, COMPUTE_R1(RR), RR, CC1, trans><<<lp.grid, lp.block, lp.shmemSize, stream>>>( \
				x.data_ptr<scalar_t>(), (const int16_t*)activeBlockIndices.data_ptr<int16_t>(), \
				y.data_ptr<scalar_t>(), N, H, W, C, bOffsH0, bOffsW0, bStrH, bStrW); \
		} else

	#define SIZE_TEMPLATES(transt, CCC) \
		CALL( 1, CCC, transt) \
		CALL( 2, CCC, transt) \
		CALL( 3, CCC, transt) \
		CALL( 4, CCC, transt) \
		CALL( 5, CCC, transt) \
		CALL( 6, CCC, transt) \
		CALL( 7, CCC, transt) \
		CALL( 8, CCC, transt) \
		CALL( 9, CCC, transt) \
		CALL(10, CCC, transt) \
		CALL(11, CCC, transt) \
		CALL(12, CCC, transt) \
		CALL(13, CCC, transt) \
		CALL(14, CCC, transt) \
		CALL(15, CCC, transt) \
		CALL(16, CCC, transt) \
		CALL(17, CCC, transt) \
		CALL(18, CCC, transt) \
		CALL(19, CCC, transt) \
		CALL(20, CCC, transt) \
		CALL(21, CCC, transt) \
		CALL(22, CCC, transt) \
		CALL(23, CCC, transt) \
		CALL(24, CCC, transt) \
		CALL(25, CCC, transt) \
		CALL(26, CCC, transt) \
		CALL(27, CCC, transt) \
		CALL(28, CCC, transt) \
		CALL(29, CCC, transt) \
		CALL(30, CCC, transt) \
		CALL(31, CCC, transt) \
		CALL(32, CCC, transt) \
		CALL(33, CCC, transt) \
		CALL(34, CCC, transt) \
		CALL(41, CCC, transt) \
		CALL(48, CCC, transt) \
		CALL(63, CCC, transt) \
		CALL(64, CCC, transt) \
		CALL(65, CCC, transt) \
		CALL(81, CCC, transt) \
		   { hasInst = false; }

	AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "SparseGatherGPU", ([&] {
		LaunchParams<scalar_t> lp(C, bSzH, bSzW, numActive);
		if (transpose) {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(true, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(true, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(true, 24)
			}
		} else {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(false, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(false, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(false, 24)
			}
		}
#endif
		if (!hasInst)
		{
			//printf("gather, C, bSzH, bSzW=%d, %d, %d, fittingC1=%d\n", C, bSzH, bSzW, lp.fittingC1);
			blockGatherTiled1<scalar_t, 512><<<lp.grid, lp.block, lp.shmemSize, stream>>>(
				x.data_ptr<scalar_t>(), (const int16_t*)activeBlockIndices.data_ptr<int16_t>(),
				y.data_ptr<scalar_t>(), N, H, W, C, bOffsH0, bOffsW0, bStrH, bStrW,
				bSzH, lp.bSzH1, bSzW, lp.fittingC1, transpose);
		}

	}));
	#undef SIZE_TEMPLATES
	#undef CALL
	gpuErrorCheck( hipPeekAtLastError() );

	return y;
}


// Define the GPU implementation that launches the CUDA kernel.
//template <typename T> struct SparseScatterFunctor<GPUDevice, T> {
//    void operator()(
//        const GPUDevice& d,
torch::Tensor LaunchSparseScatterGPU(
        torch::Tensor x, int N, int H, int W, int C, torch::Tensor y,
        int bOffsH0, int bOffsW0, int bSzH, int bSzW, int bStrH, int bStrW,
        int numActive, torch::Tensor activeBlockIndices, bool add, bool transpose, bool atomic
)
{
	bool hasInst = false;
	hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
#if 1
	#define CALL(RR, CC1, addt, transt) \
		if (bSzH == RR && bSzW == RR && lp.fittingC1 == CC1 && atomic == false) { \
			hasInst = true; \
			blockScatterTiled0<scalar_t, 512, RR, COMPUTE_R1(RR), RR, CC1, addt, transt, false> \
				<<<lp.grid, lp.block, lp.shmemSize, stream>>>( \
					x.data_ptr<scalar_t>(), \
					(const int16_t*)activeBlockIndices.data_ptr<int16_t>(), \
					y.data_ptr<scalar_t>(), N, H, W, C, bOffsH0, bOffsW0, bStrH, bStrW); \
		} else

	#define SIZE_TEMPLATES(addt, transpt, CCC) \
		CALL( 1, CCC, addt, transpt) \
		CALL( 2, CCC, addt, transpt) \
		CALL( 3, CCC, addt, transpt) \
		CALL( 4, CCC, addt, transpt) \
		CALL( 5, CCC, addt, transpt) \
		CALL( 6, CCC, addt, transpt) \
		CALL( 7, CCC, addt, transpt) \
		CALL( 8, CCC, addt, transpt) \
		CALL( 9, CCC, addt, transpt) \
		CALL(10, CCC, addt, transpt) \
		CALL(11, CCC, addt, transpt) \
		CALL(12, CCC, addt, transpt) \
		CALL(13, CCC, addt, transpt) \
		CALL(14, CCC, addt, transpt) \
		CALL(15, CCC, addt, transpt) \
		CALL(16, CCC, addt, transpt) \
		CALL(17, CCC, addt, transpt) \
		CALL(18, CCC, addt, transpt) \
		CALL(19, CCC, addt, transpt) \
		CALL(20, CCC, addt, transpt) \
		CALL(21, CCC, addt, transpt) \
		CALL(22, CCC, addt, transpt) \
		CALL(23, CCC, addt, transpt) \
		CALL(24, CCC, addt, transpt) \
		CALL(25, CCC, addt, transpt) \
		CALL(26, CCC, addt, transpt) \
		CALL(27, CCC, addt, transpt) \
		CALL(28, CCC, addt, transpt) \
		CALL(29, CCC, addt, transpt) \
		CALL(30, CCC, addt, transpt) \
		CALL(31, CCC, addt, transpt) \
		CALL(32, CCC, addt, transpt) \
		CALL(33, CCC, addt, transpt) \
		CALL(34, CCC, addt, transpt) \
		CALL(41, CCC, addt, transpt) \
		CALL(48, CCC, addt, transpt) \
		CALL(63, CCC, addt, transpt) \
		CALL(64, CCC, addt, transpt) \
		CALL(65, CCC, addt, transpt) \
		CALL(81, CCC, addt, transpt) \
			hasInst = false;
	AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "SparseScatterGPU", ([&] {
		LaunchParams<scalar_t> lp(C, bSzH, bSzW, numActive);
		if (transpose && !add) {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(false, true, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(false, true, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(false, true, 24)
			}
		} else if (transpose && add) {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(true, true, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(true, true, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(true, true, 24)
			}
		} else if (!transpose && !add) {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(false, false, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(false, false, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(false, false, 24)
			}
		} else {
			if (lp.fittingC1 >= 32) {
				SIZE_TEMPLATES(true, false, 32)
			} else if (lp.fittingC1 == 16) {
				SIZE_TEMPLATES(true, false, 16)
			} else if (lp.fittingC1 == 24) {
				SIZE_TEMPLATES(true, false, 24)
			}
		}
	#endif
		if (!hasInst) {
			//printf("scatter, C, bSzH, bSzW=%d, %d, %d, fittingC1=%d\n", C, bSzH, bSzW, lp.fittingC1);
			blockScatterTiled1<scalar_t, 512><<<lp.grid, lp.block, lp.shmemSize, stream>>>(
				x.data_ptr<scalar_t>(), (const int16_t*)activeBlockIndices.data_ptr<int16_t>(),
				y.data_ptr<scalar_t>(), N, H, W, C, bOffsH0, bOffsW0, bStrH, bStrW,
				bSzH, lp.bSzH1, bSzW, lp.fittingC1, add, transpose, atomic);
		}
	}));
	#undef SIZE_TEMPLATES
	#undef CALL
	gpuErrorCheck( hipPeekAtLastError() );

	return y;
}

//template<typename T> struct CopyTensorFunctor<GPUDevice, T> {
//    void operator()(const GPUDevice& gpu, T* dst, const T* src, int count) {
//        hipMemcpyAsync(dst, src, sizeof(T)*count, hipMemcpyDeviceToDevice, gpu.stream());
//        gpuErrorCheck( hipPeekAtLastError() );
//        hipStreamSynchronize(gpu.stream());
//        gpuErrorCheck( hipPeekAtLastError() );
//    }
//    const hipStream_t* getStream(const GPUDevice& gpu) { return &gpu.stream(); }
//};
